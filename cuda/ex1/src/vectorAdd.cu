#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

// #include <hip/hip_runtime_api.h>

#include <time.h>
#include <chrono>
#include "gputimer.h"

#define EPS 1e-5

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const float *A, const float *B, float *C,
                          int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  while (i < numElements) {
    C[i] = A[i] + B[i] + 0.0f;
    i += gridDim.x;
  }
}

/**
 * @brief Add 2 vectors using CPU
 * 
 */
void cpuVectorAdd(const float* A, const float* B, float* C,
                  int numElements) {
  --numElements;
  for (; numElements >= 0; --numElements) {
    C[numElements] = A[numElements] + B[numElements] + 0.0f;
  }
}

/**
 * @brief Check results from gpu and cpu
 * 
 */
void isCpuAndGpuResultsSame(const float* cpuRes, const float* gpuResult,
                          int numElements) {
  --numElements;
  for (; numElements >= 0; --numElements) {
    if ( fabs(cpuRes[numElements] - gpuResult[numElements]) > EPS ) {
      fprintf(stderr, "Result verification failed at element %d!\n", numElements);
      exit(EXIT_FAILURE);
    }
  }
}

void printArray(const float* array, int size) {
  for (; size >= 0; --size) {
    printf ("%f ", array[size]);
  }
  printf("\n");
}


/**
 * Host main routine
 */
int main(int argc, char** argv) {
  if (argc == 1) {
    printf("Usage: %s [blockSize] [c(puCheck)] [blocks] [blockSize]", argv[0]);
  }

  // Print the vector length to be used, and compute its size
  int numElements = 50000;

  if (argc >= 2) {
    numElements = atoi(argv[1]);
  }


  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  size_t size = numElements * sizeof(float);

  // Allocate the host input vector A
  float *h_A = (float *)malloc(size);

  // Allocate the host input vector B
  float *h_B = (float *)malloc(size);

  // Allocate the host output vector C
  float *h_C = (float *)malloc(size);

  // Verify that allocations succeeded
  if (h_A == NULL || h_B == NULL || h_C == NULL) {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    exit(EXIT_FAILURE);
  }

  // Initialize the host input vectors
  for (int i = 0; i < numElements; ++i) {
    h_A[i] = rand() / (float)RAND_MAX;
    h_B[i] = rand() / (float)RAND_MAX;
  }

  // Allocate the device input vector A
  float *d_A = NULL;
  err = hipMalloc((void **)&d_A, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Allocate the device input vector B
  float *d_B = NULL;
  err = hipMalloc((void **)&d_B, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Allocate the device output vector C
  float *d_C = NULL;
  err = hipMalloc((void **)&d_C, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy the host input vectors A and B in host memory to the device input
  // vectors in
  // device memory
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector A from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector B from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Launch the Vector Add CUDA Kernel

  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

  if (argc == 5) {
    threadsPerBlock = atoi(argv[4]);
    blocksPerGrid = atoi(argv[3]);
  }

  printf("%d %d %d ", numElements, blocksPerGrid, threadsPerBlock);

  GpuTimer timer;
  timer.Start();
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
  timer.Stop();

  printf("%f ", timer.Elapsed());

  err = hipGetLastError();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector C from device to host (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  if (argc >= 3 && argv[2][0] == 'c') {
    float* cpuRes = (float *)malloc(size);

    auto start = std::chrono::steady_clock::now();
    cpuVectorAdd(h_A, h_B, cpuRes, numElements);
    auto end = std::chrono::steady_clock::now();
    float diff = 1.0 * std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count() / 1000000;

    printf("%f\n", diff);
    isCpuAndGpuResultsSame(cpuRes, h_C, numElements);
    free(cpuRes);
  }

  // Free device global memory
  err = hipFree(d_A);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_B);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_C);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}
