#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>


#include "gputimer.h"

#define BLOCK_SIZE 32 

__global__ void matrix_transpose_naive(int *input, int *output, int matrix_size) {

	const int coef_x = gridDim.x * blockDim.x;
	const int coef_y = gridDim.y * blockDim.y;

	int indexX = threadIdx.x + blockIdx.x * blockDim.x;
	int indexY = threadIdx.y + blockIdx.y * blockDim.y + coef_y * (indexX / matrix_size);
	indexX %= matrix_size;

	while (indexX < matrix_size && indexY < matrix_size) {
		int index = indexY * matrix_size + indexX;
		int transposedIndex = indexX * matrix_size + indexY;

		output[transposedIndex] = input[index];

		indexX += coef_x;
		if (indexX >= matrix_size) {
			indexY += coef_y * (indexX / matrix_size);
			indexX %= matrix_size;
		}

		index = indexY * matrix_size + indexX;
		transposedIndex = indexX * matrix_size + indexY;
	}
}

__global__ void matrix_transpose_shared(int *input, int *output, int matrix_size) {

	__shared__ int sharedMemory [BLOCK_SIZE] [BLOCK_SIZE];
	
	const int coef_x = gridDim.x * blockDim.x;
	const int coef_y = gridDim.y * blockDim.y;

	// global index	
	int indexX = threadIdx.x + blockIdx.x * blockDim.x;
	int indexY = threadIdx.y + blockIdx.y * blockDim.y + coef_y * (indexX / matrix_size);
	indexX %= matrix_size;

	// local index
	const int localIndexX = threadIdx.x;
	const int localIndexY = threadIdx.y;

	while (indexX < matrix_size && indexY < matrix_size) {
		int index = indexY * matrix_size + indexX;
		int transposedIndex = indexX * matrix_size + indexY;
		
		sharedMemory[localIndexX][localIndexY] = input[index];

		__syncthreads();

		// writing into global memory in coalesed fashion via transposed data in shared memory
		output[transposedIndex] = sharedMemory[localIndexX][localIndexY];

		indexX += coef_x;
		if (indexX >= matrix_size) {
			indexY += coef_y * (indexX / matrix_size);
			indexX %= matrix_size;
		}

		index = indexY * matrix_size + indexX;
		transposedIndex = indexX * matrix_size + indexY;
	}
}

//basically just fills the array with index.
void fill_array(int *data, int matrix_size) {
	for(int idx=0;idx<(matrix_size * matrix_size);idx++)
		data[idx] = idx;
}

bool is_transposed(int *a, int *b, int matrix_size) {
	for (int i = 0; i < matrix_size* matrix_size; ++i) {
		int row = i / matrix_size;
		int col = i % matrix_size;

		if (a[row * matrix_size + col] != b[col * matrix_size + row]) {
			printf("Wrong value at %d %d: (%d) %d\n", row, col, a[row * matrix_size + col], b[col * matrix_size + row]);
			printf("Near: %d (%d), %d (%d)\n", b[i + 1], a[i + 1], b[i + 2], a[i + 2]);
			return false;
		}
	}
	return true;
} 

void print_output(int *a, int *b, int matrix_size) {
	printf("\n Original Matrix::\n");
	for(int idx=0;idx<(matrix_size*matrix_size);idx++) {
		if(idx % matrix_size == 0)
			printf("\n");
		printf(" %d ",  a[idx]);
	}
	printf("\n Transposed Matrix::\n");
	for(int idx=0;idx<(matrix_size*matrix_size);idx++) {
		if(idx % matrix_size == 0)
			printf("\n");
		printf(" %d ",  b[idx]);
	}
	printf("\n");
}

int main(int argc, char* argv[]) {
	int *a, *b;
        int *d_a, *d_b; // device copies of a, b, c

	if (argc != 4) {
		fprintf(stderr, "Usage %s <matrix_size> <blocks> <threads_per_block>\n", argv[0]);
		exit(1);
	}

	int matrix_size = atoi(argv[1]);
	int blocks = atoi(argv[2]);
	int threads_per_block = atoi(argv[3]);

	if (threads_per_block > BLOCK_SIZE) {
		fprintf(stderr, "Max allowed threads per block: 32\n");
		exit(2);
	}

	printf("%d %d %d ", matrix_size, blocks, threads_per_block);

	int size = matrix_size * matrix_size * sizeof(int);
	hipError_t returnValue;

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); fill_array(a, matrix_size);
	b = (int *)malloc(size);

	// Alloc space for device copies of a, b, c
	returnValue = hipMalloc((void **)&d_a, size);
	// printf("%s ", hipGetErrorString(returnValue));
	returnValue = hipMalloc((void **)&d_b, size);
	// printf("%s ", hipGetErrorString(returnValue));

	// Copy inputs to device
	returnValue = hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	// printf("%s ", hipGetErrorString(returnValue));
	returnValue = hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	// printf("%s\n", hipGetErrorString(returnValue));

	dim3 blockSize(threads_per_block, threads_per_block,1);
	dim3 gridSize(blocks / 5, 5, 1);

	GpuTimer timer;
	hipError_t err = hipSuccess;

	timer.Start();
	matrix_transpose_naive<<<gridSize, blockSize>>>(d_a, d_b, matrix_size);
	timer.Stop();
	
	printf("%f ", timer.Elapsed());

	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "1: Failed to launch error kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy result back to host
	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

	if (!is_transposed(a, b, matrix_size)) {
		fprintf(stderr, "Naive transpose failed!\n");
	}
	// print_output(a, b, matrix_size);

	timer.Start();
	matrix_transpose_shared<<<gridSize, blockSize>>>(d_a, d_b, matrix_size);
	timer.Stop();

	printf("%f\n", timer.Elapsed());

	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "2: Failed to launch error kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy result back to host
	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
	if (!is_transposed(a, b, matrix_size)) {
		fprintf(stderr, "Shared transpose failed!\n");
	}

	// terminate memories
	free(a);
	free(b);
    hipFree(d_a);
	hipFree(d_b); 

	return 0;
}
