
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 2

__global__ void hello(void) {
    printf("Dim grid: %d %d %d\n", gridDim.x, gridDim.y, gridDim.z);
    printf("Hello World! from thread [%d, %d] From device\n", threadIdx.x, blockIdx.x);
}

int main() {
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();
    hipDeviceSynchronize();
    printf("ALL DONE!\n");
    return 0;
}